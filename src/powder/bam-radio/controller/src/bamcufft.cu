// -*- c++ -*-
//
// cuFFT wrapper

#include "bamcufft.h"

#include <boost/format.hpp>

#define CUDA(expr)                                                             \
  do {                                                                         \
    expr;                                                                      \
    auto err = hipGetLastError();                                             \
    if (err != hipSuccess) {                                                  \
      throw std::runtime_error((boost::format("CUDA Error (%1%:%2%): %3%") %   \
                                __FILE__ % __LINE__ % hipGetErrorString(err)) \
                                   .str());                                    \
    }                                                                          \
  } while (false)
#define CUFFT(expr)                                                            \
  do {                                                                         \
    auto err = expr;                                                           \
    if (err != HIPFFT_SUCCESS) {                                                \
      throw std::runtime_error((boost::format("CUFFT Error (%1%:%2%): %3%") %  \
                                __FILE__ % __LINE__ % err)                     \
                                   .str());                                    \
    }                                                                          \
  } while (false)

namespace bamradio {
namespace fft {
//
// GPU FFT interface (cuFFT)
//
GPUFFT::GPUFFT(std::vector<size_t> const &sizes, bool forward,
               hipStream_t stream)
    : _type(forward ? HIPFFT_FORWARD : HIPFFT_BACKWARD) {
  // compute fft plans for all requested sizes
  for (auto const &size : sizes) {
    if (_plans.count(size) > 0)
      continue;
    hipfftHandle h;
    CUFFT(hipfftPlan1d(&h, size, HIPFFT_C2C, 1));
    CUFFT(hipfftSetStream(h, stream));
    _plans[size] = h;
  }
  assert(_plans.size() == sizes.size());
}

GPUFFT::~GPUFFT() {
  for (auto &plan : _plans) {
    hipfftDestroy(plan.second);
  }
}

void GPUFFT::execute(size_t size, fcomplex *in, fcomplex *out) const {
  CUFFT(hipfftExecC2C(_plans.at(size), (hipfftComplex *)in, (hipfftComplex *)out,
                     _type));
}

std::vector<size_t> GPUFFT::sizes() const {
  decltype(sizes()) o;
  for (auto const &p : _plans) {
    o.push_back(p.first);
  }
  std::sort(begin(o), end(o));
  return o;
}

bool GPUFFT::forward() const { return _type == HIPFFT_FORWARD; }
} // namespace fft
} // namespace bamradio
